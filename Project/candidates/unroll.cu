#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define BLOCK_SIZE 16 //@@ You can change this

__constant__ float Constant_mask[3136];

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K) //const int layer
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a
    const int shared_mem_size = BLOCK_SIZE + K -1;
    extern __shared__ float shared_mem[];
    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) Constant_mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    // #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    #define shared_mem_3d(i2,i1,i0) shared_mem[ (i2) * (shared_mem_size*shared_mem_size) + (i1)*shared_mem_size + i0]
    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int blocknum_per_row = (Width_out - 1)/BLOCK_SIZE + 1; // how many features there are
    int w_out = BLOCK_SIZE * (by % blocknum_per_row) + tx; // width out for this thread would depend on remain 
    int h_out = BLOCK_SIZE * (by / blocknum_per_row) + ty; // height for this thread would depend on how many rows we've come through
    int batch_out = bz; //bz is for batches
    int feature_out = bx; //bx is for different output features

    int block_lu_x = BLOCK_SIZE * (by % blocknum_per_row);
    int block_lu_y = BLOCK_SIZE * (by / blocknum_per_row);
    if (Map_out == 4){
        for (int c = 0; c < Channel; c++){
            for(int i = ty; i < shared_mem_size; i += BLOCK_SIZE){
                for(int j = tx; j < shared_mem_size; j += BLOCK_SIZE){
                    if (block_lu_y + i < Height && block_lu_x + j < Width){
                        shared_mem_3d(c, i, j) = in_4d(batch_out, c, block_lu_y + i, block_lu_x + j);
                    }
                    
                }
            }
        }
        __syncthreads();
    
        if (h_out < Height_out && w_out < Width_out){
            float result = 0;
            result += shared_mem_3d(0, ty + 0, tx + 0) * mask_4d(feature_out, 0, 0, 0)
                    + shared_mem_3d(0, ty + 0, tx + 1) * mask_4d(feature_out, 0, 0, 1)
                    + shared_mem_3d(0, ty + 0, tx + 2) * mask_4d(feature_out, 0, 0, 2)
                    + shared_mem_3d(0, ty + 0, tx + 3) * mask_4d(feature_out, 0, 0, 3)
                    + shared_mem_3d(0, ty + 0, tx + 4) * mask_4d(feature_out, 0, 0, 4)
                    + shared_mem_3d(0, ty + 0, tx + 5) * mask_4d(feature_out, 0, 0, 5)
                    + shared_mem_3d(0, ty + 0, tx + 6) * mask_4d(feature_out, 0, 0, 6)
                    + shared_mem_3d(0, ty + 1, tx + 0) * mask_4d(feature_out, 0, 1, 0)
                    + shared_mem_3d(0, ty + 1, tx + 1) * mask_4d(feature_out, 0, 1, 1)
                    + shared_mem_3d(0, ty + 1, tx + 2) * mask_4d(feature_out, 0, 1, 2)
                    + shared_mem_3d(0, ty + 1, tx + 3) * mask_4d(feature_out, 0, 1, 3)
                    + shared_mem_3d(0, ty + 1, tx + 4) * mask_4d(feature_out, 0, 1, 4)
                    + shared_mem_3d(0, ty + 1, tx + 5) * mask_4d(feature_out, 0, 1, 5)
                    + shared_mem_3d(0, ty + 1, tx + 6) * mask_4d(feature_out, 0, 1, 6);
            result += shared_mem_3d(0, ty + 2, tx + 0) * mask_4d(feature_out, 0, 2, 0)
                    + shared_mem_3d(0, ty + 2, tx + 1) * mask_4d(feature_out, 0, 2, 1)
                    + shared_mem_3d(0, ty + 2, tx + 2) * mask_4d(feature_out, 0, 2, 2)
                    + shared_mem_3d(0, ty + 2, tx + 3) * mask_4d(feature_out, 0, 2, 3)
                    + shared_mem_3d(0, ty + 2, tx + 4) * mask_4d(feature_out, 0, 2, 4)
                    + shared_mem_3d(0, ty + 2, tx + 5) * mask_4d(feature_out, 0, 2, 5)
                    + shared_mem_3d(0, ty + 2, tx + 6) * mask_4d(feature_out, 0, 2, 6);
            result += shared_mem_3d(0, ty + 3, tx + 0) * mask_4d(feature_out, 0, 3, 0)
                    + shared_mem_3d(0, ty + 3, tx + 1) * mask_4d(feature_out, 0, 3, 1)
                    + shared_mem_3d(0, ty + 3, tx + 2) * mask_4d(feature_out, 0, 3, 2)
                    + shared_mem_3d(0, ty + 3, tx + 3) * mask_4d(feature_out, 0, 3, 3)
                    + shared_mem_3d(0, ty + 3, tx + 4) * mask_4d(feature_out, 0, 3, 4)
                    + shared_mem_3d(0, ty + 3, tx + 5) * mask_4d(feature_out, 0, 3, 5)
                    + shared_mem_3d(0, ty + 3, tx + 6) * mask_4d(feature_out, 0, 3, 6)
                    + shared_mem_3d(0, ty + 4, tx + 0) * mask_4d(feature_out, 0, 4, 0)
                    + shared_mem_3d(0, ty + 4, tx + 1) * mask_4d(feature_out, 0, 4, 1)
                    + shared_mem_3d(0, ty + 4, tx + 2) * mask_4d(feature_out, 0, 4, 2)
                    + shared_mem_3d(0, ty + 4, tx + 3) * mask_4d(feature_out, 0, 4, 3)
                    + shared_mem_3d(0, ty + 4, tx + 4) * mask_4d(feature_out, 0, 4, 4)
                    + shared_mem_3d(0, ty + 4, tx + 5) * mask_4d(feature_out, 0, 4, 5)
                    + shared_mem_3d(0, ty + 4, tx + 6) * mask_4d(feature_out, 0, 4, 6);
            result += shared_mem_3d(0, ty + 5, tx + 0) * mask_4d(feature_out, 0, 5, 0)
                    + shared_mem_3d(0, ty + 5, tx + 1) * mask_4d(feature_out, 0, 5, 1)
                    + shared_mem_3d(0, ty + 5, tx + 2) * mask_4d(feature_out, 0, 5, 2)
                    + shared_mem_3d(0, ty + 5, tx + 3) * mask_4d(feature_out, 0, 5, 3)
                    + shared_mem_3d(0, ty + 5, tx + 4) * mask_4d(feature_out, 0, 5, 4)
                    + shared_mem_3d(0, ty + 5, tx + 5) * mask_4d(feature_out, 0, 5, 5)
                    + shared_mem_3d(0, ty + 5, tx + 6) * mask_4d(feature_out, 0, 5, 6)
                    + shared_mem_3d(0, ty + 6, tx + 0) * mask_4d(feature_out, 0, 6, 0)
                    + shared_mem_3d(0, ty + 6, tx + 1) * mask_4d(feature_out, 0, 6, 1)
                    + shared_mem_3d(0, ty + 6, tx + 2) * mask_4d(feature_out, 0, 6, 2)
                    + shared_mem_3d(0, ty + 6, tx + 3) * mask_4d(feature_out, 0, 6, 3)
                    + shared_mem_3d(0, ty + 6, tx + 4) * mask_4d(feature_out, 0, 6, 4)
                    + shared_mem_3d(0, ty + 6, tx + 5) * mask_4d(feature_out, 0, 6, 5)
                    + shared_mem_3d(0, ty + 6, tx + 6) * mask_4d(feature_out, 0, 6, 6);
            out_4d(batch_out, feature_out, h_out, w_out) = result;
        }
    }
    else{
        for (int c = 0; c < Channel; c++){
            for(int i = ty; i < shared_mem_size; i += BLOCK_SIZE){
                for(int j = tx; j < shared_mem_size; j += BLOCK_SIZE){
                    if (block_lu_y + i < Height && block_lu_x + j < Width){
                        shared_mem_3d(c, i, j) = in_4d(batch_out, c, block_lu_y + i, block_lu_x + j);
                    }
                    
                }
            }
        }
        __syncthreads();
    
        if (h_out < Height_out && w_out < Width_out){
            float result = 0;
            result += shared_mem_3d(0, ty + 0, tx + 0) * mask_4d(feature_out, 0, 0, 0)
                    + shared_mem_3d(0, ty + 0, tx + 1) * mask_4d(feature_out, 0, 0, 1)
                    + shared_mem_3d(0, ty + 0, tx + 2) * mask_4d(feature_out, 0, 0, 2)
                    + shared_mem_3d(0, ty + 0, tx + 3) * mask_4d(feature_out, 0, 0, 3)
                    + shared_mem_3d(0, ty + 0, tx + 4) * mask_4d(feature_out, 0, 0, 4)
                    + shared_mem_3d(0, ty + 0, tx + 5) * mask_4d(feature_out, 0, 0, 5)
                    + shared_mem_3d(0, ty + 0, tx + 6) * mask_4d(feature_out, 0, 0, 6)
                    + shared_mem_3d(0, ty + 1, tx + 0) * mask_4d(feature_out, 0, 1, 0)
                    + shared_mem_3d(0, ty + 1, tx + 1) * mask_4d(feature_out, 0, 1, 1)
                    + shared_mem_3d(0, ty + 1, tx + 2) * mask_4d(feature_out, 0, 1, 2)
                    + shared_mem_3d(0, ty + 1, tx + 3) * mask_4d(feature_out, 0, 1, 3)
                    + shared_mem_3d(0, ty + 1, tx + 4) * mask_4d(feature_out, 0, 1, 4)
                    + shared_mem_3d(0, ty + 1, tx + 5) * mask_4d(feature_out, 0, 1, 5)
                    + shared_mem_3d(0, ty + 1, tx + 6) * mask_4d(feature_out, 0, 1, 6);
            result += shared_mem_3d(0, ty + 2, tx + 0) * mask_4d(feature_out, 0, 2, 0)
                    + shared_mem_3d(0, ty + 2, tx + 1) * mask_4d(feature_out, 0, 2, 1)
                    + shared_mem_3d(0, ty + 2, tx + 2) * mask_4d(feature_out, 0, 2, 2)
                    + shared_mem_3d(0, ty + 2, tx + 3) * mask_4d(feature_out, 0, 2, 3)
                    + shared_mem_3d(0, ty + 2, tx + 4) * mask_4d(feature_out, 0, 2, 4)
                    + shared_mem_3d(0, ty + 2, tx + 5) * mask_4d(feature_out, 0, 2, 5)
                    + shared_mem_3d(0, ty + 2, tx + 6) * mask_4d(feature_out, 0, 2, 6);
            result += shared_mem_3d(0, ty + 3, tx + 0) * mask_4d(feature_out, 0, 3, 0)
                    + shared_mem_3d(0, ty + 3, tx + 1) * mask_4d(feature_out, 0, 3, 1)
                    + shared_mem_3d(0, ty + 3, tx + 2) * mask_4d(feature_out, 0, 3, 2)
                    + shared_mem_3d(0, ty + 3, tx + 3) * mask_4d(feature_out, 0, 3, 3)
                    + shared_mem_3d(0, ty + 3, tx + 4) * mask_4d(feature_out, 0, 3, 4)
                    + shared_mem_3d(0, ty + 3, tx + 5) * mask_4d(feature_out, 0, 3, 5)
                    + shared_mem_3d(0, ty + 3, tx + 6) * mask_4d(feature_out, 0, 3, 6)
                    + shared_mem_3d(0, ty + 4, tx + 0) * mask_4d(feature_out, 0, 4, 0)
                    + shared_mem_3d(0, ty + 4, tx + 1) * mask_4d(feature_out, 0, 4, 1)
                    + shared_mem_3d(0, ty + 4, tx + 2) * mask_4d(feature_out, 0, 4, 2)
                    + shared_mem_3d(0, ty + 4, tx + 3) * mask_4d(feature_out, 0, 4, 3)
                    + shared_mem_3d(0, ty + 4, tx + 4) * mask_4d(feature_out, 0, 4, 4)
                    + shared_mem_3d(0, ty + 4, tx + 5) * mask_4d(feature_out, 0, 4, 5)
                    + shared_mem_3d(0, ty + 4, tx + 6) * mask_4d(feature_out, 0, 4, 6);
            result += shared_mem_3d(0, ty + 5, tx + 0) * mask_4d(feature_out, 0, 5, 0)
                    + shared_mem_3d(0, ty + 5, tx + 1) * mask_4d(feature_out, 0, 5, 1)
                    + shared_mem_3d(0, ty + 5, tx + 2) * mask_4d(feature_out, 0, 5, 2)
                    + shared_mem_3d(0, ty + 5, tx + 3) * mask_4d(feature_out, 0, 5, 3)
                    + shared_mem_3d(0, ty + 5, tx + 4) * mask_4d(feature_out, 0, 5, 4)
                    + shared_mem_3d(0, ty + 5, tx + 5) * mask_4d(feature_out, 0, 5, 5)
                    + shared_mem_3d(0, ty + 5, tx + 6) * mask_4d(feature_out, 0, 5, 6)
                    + shared_mem_3d(0, ty + 6, tx + 0) * mask_4d(feature_out, 0, 6, 0)
                    + shared_mem_3d(0, ty + 6, tx + 1) * mask_4d(feature_out, 0, 6, 1)
                    + shared_mem_3d(0, ty + 6, tx + 2) * mask_4d(feature_out, 0, 6, 2)
                    + shared_mem_3d(0, ty + 6, tx + 3) * mask_4d(feature_out, 0, 6, 3)
                    + shared_mem_3d(0, ty + 6, tx + 4) * mask_4d(feature_out, 0, 6, 4)
                    + shared_mem_3d(0, ty + 6, tx + 5) * mask_4d(feature_out, 0, 6, 5)
                    + shared_mem_3d(0, ty + 6, tx + 6) * mask_4d(feature_out, 0, 6, 6);
            result += shared_mem_3d(1, ty + 0, tx + 0) * mask_4d(feature_out, 1, 0, 0)
                    + shared_mem_3d(1, ty + 0, tx + 1) * mask_4d(feature_out, 1, 0, 1)
                    + shared_mem_3d(1, ty + 0, tx + 2) * mask_4d(feature_out, 1, 0, 2)
                    + shared_mem_3d(1, ty + 0, tx + 3) * mask_4d(feature_out, 1, 0, 3)
                    + shared_mem_3d(1, ty + 0, tx + 4) * mask_4d(feature_out, 1, 0, 4)
                    + shared_mem_3d(1, ty + 0, tx + 5) * mask_4d(feature_out, 1, 0, 5)
                    + shared_mem_3d(1, ty + 0, tx + 6) * mask_4d(feature_out, 1, 0, 6)
                    + shared_mem_3d(1, ty + 1, tx + 0) * mask_4d(feature_out, 1, 1, 0)
                    + shared_mem_3d(1, ty + 1, tx + 1) * mask_4d(feature_out, 1, 1, 1)
                    + shared_mem_3d(1, ty + 1, tx + 2) * mask_4d(feature_out, 1, 1, 2)
                    + shared_mem_3d(1, ty + 1, tx + 3) * mask_4d(feature_out, 1, 1, 3)
                    + shared_mem_3d(1, ty + 1, tx + 4) * mask_4d(feature_out, 1, 1, 4)
                    + shared_mem_3d(1, ty + 1, tx + 5) * mask_4d(feature_out, 1, 1, 5)
                    + shared_mem_3d(1, ty + 1, tx + 6) * mask_4d(feature_out, 1, 1, 6);
            result += shared_mem_3d(1, ty + 2, tx + 0) * mask_4d(feature_out, 1, 2, 0)
                    + shared_mem_3d(1, ty + 2, tx + 1) * mask_4d(feature_out, 1, 2, 1)
                    + shared_mem_3d(1, ty + 2, tx + 2) * mask_4d(feature_out, 1, 2, 2)
                    + shared_mem_3d(1, ty + 2, tx + 3) * mask_4d(feature_out, 1, 2, 3)
                    + shared_mem_3d(1, ty + 2, tx + 4) * mask_4d(feature_out, 1, 2, 4)
                    + shared_mem_3d(1, ty + 2, tx + 5) * mask_4d(feature_out, 1, 2, 5)
                    + shared_mem_3d(1, ty + 2, tx + 6) * mask_4d(feature_out, 1, 2, 6)
                    + shared_mem_3d(1, ty + 3, tx + 0) * mask_4d(feature_out, 1, 3, 0)
                    + shared_mem_3d(1, ty + 3, tx + 1) * mask_4d(feature_out, 1, 3, 1)
                    + shared_mem_3d(1, ty + 3, tx + 2) * mask_4d(feature_out, 1, 3, 2)
                    + shared_mem_3d(1, ty + 3, tx + 3) * mask_4d(feature_out, 1, 3, 3)
                    + shared_mem_3d(1, ty + 3, tx + 4) * mask_4d(feature_out, 1, 3, 4)
                    + shared_mem_3d(1, ty + 3, tx + 5) * mask_4d(feature_out, 1, 3, 5)
                    + shared_mem_3d(1, ty + 3, tx + 6) * mask_4d(feature_out, 1, 3, 6);
            result += shared_mem_3d(2, ty + 0, tx + 0) * mask_4d(feature_out, 2, 0, 0)
                    + shared_mem_3d(2, ty + 0, tx + 1) * mask_4d(feature_out, 2, 0, 1)
                    + shared_mem_3d(2, ty + 0, tx + 2) * mask_4d(feature_out, 2, 0, 2)
                    + shared_mem_3d(2, ty + 0, tx + 3) * mask_4d(feature_out, 2, 0, 3)
                    + shared_mem_3d(2, ty + 0, tx + 4) * mask_4d(feature_out, 2, 0, 4)
                    + shared_mem_3d(2, ty + 0, tx + 5) * mask_4d(feature_out, 2, 0, 5)
                    + shared_mem_3d(2, ty + 0, tx + 6) * mask_4d(feature_out, 2, 0, 6)
                    + shared_mem_3d(2, ty + 1, tx + 0) * mask_4d(feature_out, 2, 1, 0)
                    + shared_mem_3d(2, ty + 1, tx + 1) * mask_4d(feature_out, 2, 1, 1)
                    + shared_mem_3d(2, ty + 1, tx + 2) * mask_4d(feature_out, 2, 1, 2)
                    + shared_mem_3d(2, ty + 1, tx + 3) * mask_4d(feature_out, 2, 1, 3)
                    + shared_mem_3d(2, ty + 1, tx + 4) * mask_4d(feature_out, 2, 1, 4)
                    + shared_mem_3d(2, ty + 1, tx + 5) * mask_4d(feature_out, 2, 1, 5)
                    + shared_mem_3d(2, ty + 1, tx + 6) * mask_4d(feature_out, 2, 1, 6);
            result += shared_mem_3d(2, ty + 2, tx + 0) * mask_4d(feature_out, 2, 2, 0)
                    + shared_mem_3d(2, ty + 2, tx + 1) * mask_4d(feature_out, 2, 2, 1)
                    + shared_mem_3d(2, ty + 2, tx + 2) * mask_4d(feature_out, 2, 2, 2)
                    + shared_mem_3d(2, ty + 2, tx + 3) * mask_4d(feature_out, 2, 2, 3)
                    + shared_mem_3d(2, ty + 2, tx + 4) * mask_4d(feature_out, 2, 2, 4)
                    + shared_mem_3d(2, ty + 2, tx + 5) * mask_4d(feature_out, 2, 2, 5)
                    + shared_mem_3d(2, ty + 2, tx + 6) * mask_4d(feature_out, 2, 2, 6)
                    + shared_mem_3d(2, ty + 3, tx + 0) * mask_4d(feature_out, 2, 3, 0)
                    + shared_mem_3d(2, ty + 3, tx + 1) * mask_4d(feature_out, 2, 3, 1)
                    + shared_mem_3d(2, ty + 3, tx + 2) * mask_4d(feature_out, 2, 3, 2)
                    + shared_mem_3d(2, ty + 3, tx + 3) * mask_4d(feature_out, 2, 3, 3)
                    + shared_mem_3d(2, ty + 3, tx + 4) * mask_4d(feature_out, 2, 3, 4)
                    + shared_mem_3d(2, ty + 3, tx + 5) * mask_4d(feature_out, 2, 3, 5)
                    + shared_mem_3d(2, ty + 3, tx + 6) * mask_4d(feature_out, 2, 3, 6);
            result += shared_mem_3d(2, ty + 4, tx + 0) * mask_4d(feature_out, 2, 4, 0)
                    + shared_mem_3d(2, ty + 4, tx + 1) * mask_4d(feature_out, 2, 4, 1)
                    + shared_mem_3d(2, ty + 4, tx + 2) * mask_4d(feature_out, 2, 4, 2)
                    + shared_mem_3d(2, ty + 4, tx + 3) * mask_4d(feature_out, 2, 4, 3)
                    + shared_mem_3d(2, ty + 4, tx + 4) * mask_4d(feature_out, 2, 4, 4)
                    + shared_mem_3d(2, ty + 4, tx + 5) * mask_4d(feature_out, 2, 4, 5)
                    + shared_mem_3d(2, ty + 4, tx + 6) * mask_4d(feature_out, 2, 4, 6)
                    + shared_mem_3d(2, ty + 5, tx + 0) * mask_4d(feature_out, 2, 5, 0)
                    + shared_mem_3d(2, ty + 5, tx + 1) * mask_4d(feature_out, 2, 5, 1)
                    + shared_mem_3d(2, ty + 5, tx + 2) * mask_4d(feature_out, 2, 5, 2)
                    + shared_mem_3d(2, ty + 5, tx + 3) * mask_4d(feature_out, 2, 5, 3)
                    + shared_mem_3d(2, ty + 5, tx + 4) * mask_4d(feature_out, 2, 5, 4)
                    + shared_mem_3d(2, ty + 5, tx + 5) * mask_4d(feature_out, 2, 5, 5)
                    + shared_mem_3d(2, ty + 5, tx + 6) * mask_4d(feature_out, 2, 5, 6);
            result += shared_mem_3d(2, ty + 6, tx + 0) * mask_4d(feature_out, 2, 6, 0)
                    + shared_mem_3d(2, ty + 6, tx + 1) * mask_4d(feature_out, 2, 6, 1)
                    + shared_mem_3d(2, ty + 6, tx + 2) * mask_4d(feature_out, 2, 6, 2)
                    + shared_mem_3d(2, ty + 6, tx + 3) * mask_4d(feature_out, 2, 6, 3)
                    + shared_mem_3d(2, ty + 6, tx + 4) * mask_4d(feature_out, 2, 6, 4)
                    + shared_mem_3d(2, ty + 6, tx + 5) * mask_4d(feature_out, 2, 6, 5)
                    + shared_mem_3d(2, ty + 6, tx + 6) * mask_4d(feature_out, 2, 6, 6);
            result += shared_mem_3d(3, ty + 0, tx + 0) * mask_4d(feature_out, 3, 0, 0)
                    + shared_mem_3d(3, ty + 0, tx + 1) * mask_4d(feature_out, 3, 0, 1)
                    + shared_mem_3d(3, ty + 0, tx + 2) * mask_4d(feature_out, 3, 0, 2)
                    + shared_mem_3d(3, ty + 0, tx + 3) * mask_4d(feature_out, 3, 0, 3)
                    + shared_mem_3d(3, ty + 0, tx + 4) * mask_4d(feature_out, 3, 0, 4)
                    + shared_mem_3d(3, ty + 0, tx + 5) * mask_4d(feature_out, 3, 0, 5)
                    + shared_mem_3d(3, ty + 0, tx + 6) * mask_4d(feature_out, 3, 0, 6)
                    + shared_mem_3d(3, ty + 1, tx + 0) * mask_4d(feature_out, 3, 1, 0)
                    + shared_mem_3d(3, ty + 1, tx + 1) * mask_4d(feature_out, 3, 1, 1)
                    + shared_mem_3d(3, ty + 1, tx + 2) * mask_4d(feature_out, 3, 1, 2)
                    + shared_mem_3d(3, ty + 1, tx + 3) * mask_4d(feature_out, 3, 1, 3)
                    + shared_mem_3d(3, ty + 1, tx + 4) * mask_4d(feature_out, 3, 1, 4)
                    + shared_mem_3d(3, ty + 1, tx + 5) * mask_4d(feature_out, 3, 1, 5)
                    + shared_mem_3d(3, ty + 1, tx + 6) * mask_4d(feature_out, 3, 1, 6);
            result += shared_mem_3d(3, ty + 2, tx + 0) * mask_4d(feature_out, 3, 2, 0)
                    + shared_mem_3d(3, ty + 2, tx + 1) * mask_4d(feature_out, 3, 2, 1)
                    + shared_mem_3d(3, ty + 2, tx + 2) * mask_4d(feature_out, 3, 2, 2)
                    + shared_mem_3d(3, ty + 2, tx + 3) * mask_4d(feature_out, 3, 2, 3)
                    + shared_mem_3d(3, ty + 2, tx + 4) * mask_4d(feature_out, 3, 2, 4)
                    + shared_mem_3d(3, ty + 2, tx + 5) * mask_4d(feature_out, 3, 2, 5)
                    + shared_mem_3d(3, ty + 2, tx + 6) * mask_4d(feature_out, 3, 2, 6)
                    + shared_mem_3d(3, ty + 3, tx + 0) * mask_4d(feature_out, 3, 3, 0)
                    + shared_mem_3d(3, ty + 3, tx + 1) * mask_4d(feature_out, 3, 3, 1)
                    + shared_mem_3d(3, ty + 3, tx + 2) * mask_4d(feature_out, 3, 3, 2)
                    + shared_mem_3d(3, ty + 3, tx + 3) * mask_4d(feature_out, 3, 3, 3)
                    + shared_mem_3d(3, ty + 3, tx + 4) * mask_4d(feature_out, 3, 3, 4)
                    + shared_mem_3d(3, ty + 3, tx + 5) * mask_4d(feature_out, 3, 3, 5)
                    + shared_mem_3d(3, ty + 3, tx + 6) * mask_4d(feature_out, 3, 3, 6);
            result += shared_mem_3d(3, ty + 4, tx + 0) * mask_4d(feature_out, 3, 4, 0)
                    + shared_mem_3d(3, ty + 4, tx + 1) * mask_4d(feature_out, 3, 4, 1)
                    + shared_mem_3d(3, ty + 4, tx + 2) * mask_4d(feature_out, 3, 4, 2)
                    + shared_mem_3d(3, ty + 4, tx + 3) * mask_4d(feature_out, 3, 4, 3)
                    + shared_mem_3d(3, ty + 4, tx + 4) * mask_4d(feature_out, 3, 4, 4)
                    + shared_mem_3d(3, ty + 4, tx + 5) * mask_4d(feature_out, 3, 4, 5)
                    + shared_mem_3d(3, ty + 4, tx + 6) * mask_4d(feature_out, 3, 4, 6)
                    + shared_mem_3d(3, ty + 5, tx + 0) * mask_4d(feature_out, 3, 5, 0)
                    + shared_mem_3d(3, ty + 5, tx + 1) * mask_4d(feature_out, 3, 5, 1)
                    + shared_mem_3d(3, ty + 5, tx + 2) * mask_4d(feature_out, 3, 5, 2)
                    + shared_mem_3d(3, ty + 5, tx + 3) * mask_4d(feature_out, 3, 5, 3)
                    + shared_mem_3d(3, ty + 5, tx + 4) * mask_4d(feature_out, 3, 5, 4)
                    + shared_mem_3d(3, ty + 5, tx + 5) * mask_4d(feature_out, 3, 5, 5)
                    + shared_mem_3d(3, ty + 5, tx + 6) * mask_4d(feature_out, 3, 5, 6);
            result += shared_mem_3d(3, ty + 6, tx + 0) * mask_4d(feature_out, 3, 6, 0)
                    + shared_mem_3d(3, ty + 6, tx + 1) * mask_4d(feature_out, 3, 6, 1)
                    + shared_mem_3d(3, ty + 6, tx + 2) * mask_4d(feature_out, 3, 6, 2)
                    + shared_mem_3d(3, ty + 6, tx + 3) * mask_4d(feature_out, 3, 6, 3)
                    + shared_mem_3d(3, ty + 6, tx + 4) * mask_4d(feature_out, 3, 6, 4)
                    + shared_mem_3d(3, ty + 6, tx + 5) * mask_4d(feature_out, 3, 6, 5)
                    + shared_mem_3d(3, ty + 6, tx + 6) * mask_4d(feature_out, 3, 6, 6);
            out_4d(batch_out, feature_out, h_out, w_out) = result;
        }
    }
    
    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    hipMalloc((void **) device_output_ptr, (Batch * Map_out * (Height - K + 1) * (Width - K + 1))*sizeof(float));
    hipMalloc((void **) device_input_ptr, (Batch * Channel * Height * Width)*sizeof(float));
    // hipMalloc((void **) device_mask_ptr, (Map_out * Channel * K * K)*sizeof(float));
    hipMemcpy(*device_input_ptr, host_input, (Batch * Channel * Height * Width)*sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(*device_mask_ptr, host_mask, (Map_out * Channel * K * K)*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(Constant_mask), host_mask, (Map_out * Channel * K * K)*sizeof(float));
    printf("Kernel size:%d\n", (Map_out * Channel * K * K));
    printf("Input size:%d\n", (Batch * Channel * Height * Width));
    printf("kernel width:%d\n",K);
    // printf("Kernel size:", sizeof(host_mask));
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // std::cout<<"kernel size"<< K <<endl;
    // Set the kernel dimensions and call the kernel
    dim3 dimGrid(Map_out, ceil((float)(Height - K + 1)/BLOCK_SIZE)*ceil((float)(Width - K + 1)/BLOCK_SIZE), Batch);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    printf("Mapout:%d\n", Map_out);
    printf("Channel:%d\n", Channel);
    // const int layer = 0;
    // if 
    // we need a shared space of C*inputwidth_of_block*inputwidth_of_block
    conv_forward_kernel<<<dimGrid, dimBlock, Channel*(BLOCK_SIZE + K - 1)*(BLOCK_SIZE + K - 1)*sizeof(float)>>>(device_output, device_input, device_mask, Batch, Map_out, Channel, Height, Width, K);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    hipMemcpy(host_output, device_output, (Batch * Map_out * (Height - K + 1) * (Width - K + 1))*sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
