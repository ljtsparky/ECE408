#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#define BLOCK_SIZE 16 //@@ You can change this

__constant__ float Constant_mask[6000];

__global__ void conv_forward_kernel(float *output, const float *input, const float *mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    Batch - batch_size (number of images in x)
    Map_out - number of output feature maps
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (Map_out * Height_out * Width_out) + (i2) * (Height_out * Width_out) + (i1) * (Width_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define mask_4d(i3, i2, i1, i0) Constant_mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]
    // #define mask_4d(i3, i2, i1, i0) mask[(i3) * (Channel * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;

    int blocknum_per_row = (Width_out - 1)/BLOCK_SIZE + 1; // how many features there are
    int w_out = BLOCK_SIZE * (by % blocknum_per_row) + tx; // width out for this thread would depend on remain 
    int h_out = BLOCK_SIZE * (by / blocknum_per_row) + ty; // height for this thread would depend on how many rows we've come through
    int batch_out = bz; //bz is for batches
    int feature_out = bx; //bx is for different output features

    if (h_out < Height_out && w_out < Width_out){
        float result = 0;
        for (int c = 0; c < Channel; c++){
            for (int ky = 0; ky < K; ky++){
                for (int kx = 0; kx < K; kx++){
                    result += in_4d(batch_out, c, h_out + ky, w_out + kx) * mask_4d(feature_out, c, ky, kx);
                }
            }
        }
        out_4d(batch_out, feature_out, h_out, w_out) = result;
    }
    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    hipMalloc((void **) device_output_ptr, (Batch * Map_out * (Height - K + 1) * (Width - K + 1))*sizeof(float));
    hipMalloc((void **) device_input_ptr, (Batch * Channel * Height * Width)*sizeof(float));
    // // hipMalloc((void **) device_mask_ptr, (Map_out * Channel * K * K)*sizeof(float));
    // hipMemcpy(*device_input_ptr_ptr, host_input, (Batch * Channel * Height * Width)*sizeof(float), hipMemcpyHostToDevice);
    // // hipMemcpy(*device_mask_ptr, host_mask, (Map_out * Channel * K * K)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(Constant_mask), host_mask, (Map_out * Channel * K * K)*sizeof(float));
    hipStream_t stream0, stream1, stream2, stream3, stream4, stream5, stream6, stream7, stream8, stream9; //stream10, stream11; //stream12, stream13, stream14, stream15;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);
    hipStreamCreate(&stream4);
    hipStreamCreate(&stream5);
    hipStreamCreate(&stream6);
    hipStreamCreate(&stream7);
    hipStreamCreate(&stream8);
    hipStreamCreate(&stream9);
    // hipStreamCreate(&stream10);
    // hipStreamCreate(&stream11);
    // hipStreamCreate(&stream12);
    // hipStreamCreate(&stream13);
    // hipStreamCreate(&stream14);
    // hipStreamCreate(&stream15);
    int SegNum = 10;
    int Input_segment_size = Channel*Height*Width;
    int Output_segment_size = Map_out*(Height-K+1)*(Width-K+1);
    dim3 dimGrid(Map_out, ceil((float)(Height - K + 1)/BLOCK_SIZE)*ceil((float)(Width - K + 1)/BLOCK_SIZE), SegNum);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    // conv_forward_kernel<<<dimGrid, dimBlock>>>(device_output_ptr, device_input_ptr, device_mask, Batch, Map_out, Channel, Height, Width, K);
    for (int i = 0; i < Batch; i += SegNum*10){
        hipMemcpyAsync(*device_input_ptr + (i + 0 * SegNum) *Input_segment_size, host_input + (i + 0 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream0);
        hipMemcpyAsync(*device_input_ptr + (i + 1 * SegNum) *Input_segment_size, host_input + (i + 1 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream1);
        hipMemcpyAsync(*device_input_ptr + (i + 2 * SegNum) *Input_segment_size, host_input + (i + 2 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream2);
        hipMemcpyAsync(*device_input_ptr + (i + 3 * SegNum) *Input_segment_size, host_input + (i + 3 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream3);
        hipMemcpyAsync(*device_input_ptr + (i + 4 * SegNum) *Input_segment_size, host_input + (i + 4 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream4);
        hipMemcpyAsync(*device_input_ptr + (i + 5 * SegNum) *Input_segment_size, host_input + (i + 5 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream5);
        hipMemcpyAsync(*device_input_ptr + (i + 6 * SegNum) *Input_segment_size, host_input + (i + 6 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream6);
        hipMemcpyAsync(*device_input_ptr + (i + 7 * SegNum) *Input_segment_size, host_input + (i + 7 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream7);
        hipMemcpyAsync(*device_input_ptr + (i + 8 * SegNum) *Input_segment_size, host_input + (i + 8 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream8);
        hipMemcpyAsync(*device_input_ptr + (i + 9 * SegNum) *Input_segment_size, host_input + (i + 9 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream9);
        // hipMemcpyAsync(*device_input_ptr + (i + 10 * SegNum) *Input_segment_size, host_input + (i + 10 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream10);
        // hipMemcpyAsync(*device_input_ptr + (i + 11 * SegNum) *Input_segment_size, host_input + (i + 11 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream11);
        // hipMemcpyAsync(*device_input_ptr + (i + 12 * SegNum) *Input_segment_size, host_input + (i + 12 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream12);
        // hipMemcpyAsync(*device_input_ptr + (i + 13 * SegNum) *Input_segment_size, host_input + (i + 13 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream13);
        // hipMemcpyAsync(*device_input_ptr + (i + 14 * SegNum) *Input_segment_size, host_input + (i + 14 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream14);
        // hipMemcpyAsync(*device_input_ptr + (i + 15 * SegNum) *Input_segment_size, host_input + (i + 15 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream15);
        
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream0>>>(*device_output_ptr + (i + 0 * SegNum) * Output_segment_size, *device_input_ptr + (i + 0 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream1>>>(*device_output_ptr + (i + 1 * SegNum) * Output_segment_size, *device_input_ptr + (i + 1 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream2>>>(*device_output_ptr + (i + 2 * SegNum) * Output_segment_size, *device_input_ptr + (i + 2 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream3>>>(*device_output_ptr + (i + 3 * SegNum) * Output_segment_size, *device_input_ptr + (i + 3 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream4>>>(*device_output_ptr + (i + 4 * SegNum) * Output_segment_size, *device_input_ptr + (i + 4 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream5>>>(*device_output_ptr + (i + 5 * SegNum) * Output_segment_size, *device_input_ptr + (i + 5 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream6>>>(*device_output_ptr + (i + 6 * SegNum) * Output_segment_size, *device_input_ptr + (i + 6 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream7>>>(*device_output_ptr + (i + 7 * SegNum) * Output_segment_size, *device_input_ptr + (i + 7 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream8>>>(*device_output_ptr + (i + 8 * SegNum) * Output_segment_size, *device_input_ptr + (i + 8 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        conv_forward_kernel<<<dimGrid, dimBlock, 0, stream9>>>(*device_output_ptr + (i + 9 * SegNum) * Output_segment_size, *device_input_ptr + (i + 9 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        // conv_forward_kernel<<<dimGrid, dimBlock, 0, stream10>>>(*device_output_ptr + (i + 10 * SegNum) * Output_segment_size, *device_input_ptr + (i + 10 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        // conv_forward_kernel<<<dimGrid, dimBlock, 0, stream11>>>(*device_output_ptr + (i + 11 * SegNum) * Output_segment_size, *device_input_ptr + (i + 11 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        // conv_forward_kernel<<<dimGrid, dimBlock, 0, stream4>>>(*device_output_ptr + (i + 12 * SegNum) * Output_segment_size, *device_input_ptr + (i + 12 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        // conv_forward_kernel<<<dimGrid, dimBlock, 0, stream5>>>(*device_output_ptr + (i + 13 * SegNum) * Output_segment_size, *device_input_ptr + (i + 13 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        // conv_forward_kernel<<<dimGrid, dimBlock, 0, stream6>>>(*device_output_ptr + (i + 14 * SegNum) * Output_segment_size, *device_input_ptr + (i + 14 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
        // conv_forward_kernel<<<dimGrid, dimBlock, 0, stream7>>>(*device_output_ptr + (i + 15 * SegNum) * Output_segment_size, *device_input_ptr + (i + 15 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);

        hipMemcpyAsync(host_output + (i + 0 * SegNum) * Output_segment_size, *device_output_ptr + (i + 0 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream0);
        hipMemcpyAsync(host_output + (i + 1 * SegNum) * Output_segment_size, *device_output_ptr + (i + 1 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream1);
        hipMemcpyAsync(host_output + (i + 2 * SegNum) * Output_segment_size, *device_output_ptr + (i + 2 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream2);
        hipMemcpyAsync(host_output + (i + 3 * SegNum) * Output_segment_size, *device_output_ptr + (i + 3 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream3);
        hipMemcpyAsync(host_output + (i + 4 * SegNum) * Output_segment_size, *device_output_ptr + (i + 4 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream4);
        hipMemcpyAsync(host_output + (i + 5 * SegNum) * Output_segment_size, *device_output_ptr + (i + 5 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream5);
        hipMemcpyAsync(host_output + (i + 6 * SegNum) * Output_segment_size, *device_output_ptr + (i + 6 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream6);
        hipMemcpyAsync(host_output + (i + 7 * SegNum) * Output_segment_size, *device_output_ptr + (i + 7 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream7);
        hipMemcpyAsync(host_output + (i + 8 * SegNum) * Output_segment_size, *device_output_ptr + (i + 8 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream8);
        hipMemcpyAsync(host_output + (i + 9 * SegNum) * Output_segment_size, *device_output_ptr + (i + 9 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream9);
        // hipMemcpyAsync(host_output + (i + 10 * SegNum) * Output_segment_size, *device_output_ptr + (i + 10 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream10);
        // hipMemcpyAsync(host_output + (i + 11 * SegNum) * Output_segment_size, *device_output_ptr + (i + 11 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream11);
        // hipMemcpyAsync(host_output + (i + 12 * SegNum) * Output_segment_size, *device_output_ptr + (i + 12 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream12);
        // hipMemcpyAsync(host_output + (i + 13 * SegNum) * Output_segment_size, *device_output_ptr + (i + 13 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream5);
        // hipMemcpyAsync(host_output + (i + 14 * SegNum) * Output_segment_size, *device_output_ptr + (i + 14 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream6);
        // hipMemcpyAsync(host_output + (i + 15 * SegNum) * Output_segment_size, *device_output_ptr + (i + 15 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream7);
    }
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output_ptr, const float *device_input_ptr, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{

    // std::cout<<"kernel size"<< K <<endl;
    // Set the kernel dimensions and call the kernel
    // int Input_segment_size = Channel*Height*Width;
    // int Output_segment_size = Map_out*(Height-K+1)*(Width-K+1)
    // dim3 dimGrid(Map_out, ceil((float)(Height - K + 1)/BLOCK_SIZE)*ceil((float)(Width - K + 1)/BLOCK_SIZE), SegNum);
    // dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
    // // conv_forward_kernel<<<dimGrid, dimBlock>>>(device_output_ptr, device_input_ptr, device_mask, Batch, Map_out, Channel, Height, Width, K);
    // for (int i = 0; i < Batch; i += SegNum*8){
    //     hipMemcpyAsync(*device_input_ptr + (i + 0 * SegNum) *Input_segment_size, host_input + (i + 0 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream0);
    //     hipMemcpyAsync(*device_input_ptr + (i + 1 * SegNum) *Input_segment_size, host_input + (i + 1 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream1);
    //     hipMemcpyAsync(*device_input_ptr + (i + 2 * SegNum) *Input_segment_size, host_input + (i + 2 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream2);
    //     hipMemcpyAsync(*device_input_ptr + (i + 3 * SegNum) *Input_segment_size, host_input + (i + 3 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream3);
    //     hipMemcpyAsync(*device_input_ptr + (i + 4 * SegNum) *Input_segment_size, host_input + (i + 4 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream4);
    //     hipMemcpyAsync(*device_input_ptr + (i + 5 * SegNum) *Input_segment_size, host_input + (i + 5 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream5);
    //     hipMemcpyAsync(*device_input_ptr + (i + 6 * SegNum) *Input_segment_size, host_input + (i + 6 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream6);
    //     hipMemcpyAsync(*device_input_ptr + (i + 7 * SegNum) *Input_segment_size, host_input + (i + 7 * SegNum) *Input_segment_size, SegNum *Input_segment_size * sizeof(float), hipMemcpyHostToDevice, stream7);

    //     conv_forward_kernel<<<dimGrid, dimBlock, 0, stream0>>>(*device_output_ptr + (i + 0 * SegNum) * Output_segment_size, *device_input_ptr + (i + 0 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
    //     conv_forward_kernel<<<dimGrid, dimBlock, 0, stream1>>>(*device_output_ptr + (i + 1 * SegNum) * Output_segment_size, *device_input_ptr + (i + 1 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
    //     conv_forward_kernel<<<dimGrid, dimBlock, 0, stream2>>>(*device_output_ptr + (i + 2 * SegNum) * Output_segment_size, *device_input_ptr + (i + 2 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
    //     conv_forward_kernel<<<dimGrid, dimBlock, 0, stream3>>>(*device_output_ptr + (i + 3 * SegNum) * Output_segment_size, *device_input_ptr + (i + 3 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
    //     conv_forward_kernel<<<dimGrid, dimBlock, 0, stream4>>>(*device_output_ptr + (i + 4 * SegNum) * Output_segment_size, *device_input_ptr + (i + 4 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
    //     conv_forward_kernel<<<dimGrid, dimBlock, 0, stream5>>>(*device_output_ptr + (i + 5 * SegNum) * Output_segment_size, *device_input_ptr + (i + 5 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
    //     conv_forward_kernel<<<dimGrid, dimBlock, 0, stream6>>>(*device_output_ptr + (i + 6 * SegNum) * Output_segment_size, *device_input_ptr + (i + 6 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);
    //     conv_forward_kernel<<<dimGrid, dimBlock, 0, stream7>>>(*device_output_ptr + (i + 7 * SegNum) * Output_segment_size, *device_input_ptr + (i + 7 * SegNum) *Input_segment_size, *device_mask_ptr, Batch, Map_out, Channel, Height, Width, K);

    //     hipMemcpyAsync(host_input + (i + 0 * SegNum) * Output_segment_size, *device_output_ptr + (i + 0 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream0);
    //     hipMemcpyAsync(host_input + (i + 1 * SegNum) * Output_segment_size, *device_output_ptr + (i + 1 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream1);
    //     hipMemcpyAsync(host_input + (i + 2 * SegNum) * Output_segment_size, *device_output_ptr + (i + 2 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream2);
    //     hipMemcpyAsync(host_input + (i + 3 * SegNum) * Output_segment_size, *device_output_ptr + (i + 3 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream3);
    //     hipMemcpyAsync(host_input + (i + 4 * SegNum) * Output_segment_size, *device_output_ptr + (i + 4 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream4);
    //     hipMemcpyAsync(host_input + (i + 5 * SegNum) * Output_segment_size, *device_output_ptr + (i + 5 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream5);
    //     hipMemcpyAsync(host_input + (i + 6 * SegNum) * Output_segment_size, *device_output_ptr + (i + 6 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream6);
    //     hipMemcpyAsync(host_input + (i + 7 * SegNum) * Output_segment_size, *device_output_ptr + (i + 7 * SegNum) * Output_segment_size, SegNum * Output_segment_size * sizeof(float), hipMemcpyDeviceToHost, stream7);
    // }
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output_ptr, float *device_input_ptr, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // Copy the output back to host
    // hipMemcpy(host_output, device_output_ptr, (Batch * Map_out * (Height - K + 1) * (Width - K + 1))*sizeof(float), hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(device_output_ptr);
    hipFree(device_input_ptr);
    // hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
